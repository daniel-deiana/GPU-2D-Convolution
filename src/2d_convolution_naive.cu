
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>

// baseline 2d convolution
// Only use odd kernel sizes
__global__ void convolution_2d(int *A,int *F, int p, int n, int *C) {

  int tmp = 0;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = blockIdx.x*blockDim.x + tx;
  int col = blockIdx.y*blockDim.y + ty;

  int offset_k = p/2;

  int row_offset = row - offset_k; 
  int col_offset = col - offset_k;

  for(int kx = 0; kx < p; kx++) {
    for(int ky = 0; ky < p; ky++) {

      if(row_offset + kx >= 0 && row_offset + kx < n) {
        if(col_offset + ky >= 0 && col_offset + ky < n)
          tmp += A[(row_offset + kx)*n + col_offset + ky] 
                * F[kx * p + ky];
      }
    }
  }

  C[row*n + col] = tmp;
}

int main () {

  const int WARP_SIZE = 32;

  const int N = 4;
  const int P = 3;

  int A[N][N], C[N][N], F[P][P];
  int *c_A, *c_F, *c_C;

  int a_size = sizeof(int) *N*N;
  int f_size = sizeof(int) *P*P;

  // allocate memory on device
  hipMalloc((void**)&c_A,a_size);
  hipMalloc((void**)&c_F,f_size);
  hipMalloc((void**)&c_C,a_size);

  for(int j=0; j < N; j++) {
    for(int k=0; k < N; k++) { A[j][k] = 1; C[j][k] = 0; }}

  for(int j=0; j < P; j++) {
          for(int k=0; k < P; k++) { A[j][k] = 1; }}

  hipMemcpy(c_A,&A,a_size,hipMemcpyHostToDevice);
  hipMemcpy(c_F,&F,f_size,hipMemcpyHostToDevice);
  hipMemcpy(c_C,&C,a_size,hipMemcpyHostToDevice);

  dim3 block(16,16);

  convolution_2d<<<1,block>>>(c_A, c_F, P, N, c_C);

  // transfer back on host from device
  hipMemcpy(C,c_C,a_size,hipMemcpyDeviceToHost);

  for(int j=0; j < N; j++) {
    for(int k=0; k < N; k++) {
      std::cout << C[j][k];
    }
    std::cout << std::endl;
  }


}
